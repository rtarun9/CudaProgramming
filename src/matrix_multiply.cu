
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define MATRIX_DIMENSION 32

// Idea on how to approach.
// Consider the matrix to be 32x32.
// 1 2 3 4 5 .... 31
// 32 33 ......   63
// ........     1023

// We can solve the problem using a single block on the block grid consisting of 32 x 32 (1024) threads.
// So, blockDim = (1, 1, 1), and threadGroupDim = (32, 32, 1)

__global__ void matrix_multiply(const int *const matrix_a, const int *const matrix_b, int *const matrix_sum_result)
{
    const int i = threadIdx.x + blockIdx.x * blockDim.x;
    const int j = threadIdx.y + blockIdx.y * blockDim.y;

    const int index = i + j * MATRIX_DIMENSION;

    int accumulator = 0;

    for (int k = 0; k < MATRIX_DIMENSION; k++)
    {
        // Copy data from global memory to registers.
        int a = matrix_a[k + i * MATRIX_DIMENSION];
        int b = matrix_b[k * MATRIX_DIMENSION + j];

        accumulator += a * b;
    }

    matrix_sum_result[index] = accumulator;
}

int main()
{
    // Setup host side data / buffers.
    int *host_matrix_a = NULL;
    int *host_matrix_b = NULL;
    int *host_matrix_product_result = NULL;

    host_matrix_a = (int *)malloc(sizeof(int) * MATRIX_DIMENSION * MATRIX_DIMENSION);
    host_matrix_b = (int *)malloc(sizeof(int) * MATRIX_DIMENSION * MATRIX_DIMENSION);
    host_matrix_product_result = (int *)calloc(MATRIX_DIMENSION * MATRIX_DIMENSION, sizeof(int));

    for (int i = 0; i < MATRIX_DIMENSION * MATRIX_DIMENSION; i++)
    {
        host_matrix_a[i] = i;
        host_matrix_b[i] = i;
    }

    // Setup device side data / buffers.
    int *device_matrix_a = NULL;
    int *device_matrix_b = NULL;
    int *device_matrix_product_result = NULL;

    hipMalloc((void **)&device_matrix_a, MATRIX_DIMENSION * MATRIX_DIMENSION * sizeof(int));
    hipMalloc((void **)&device_matrix_b, MATRIX_DIMENSION * MATRIX_DIMENSION * sizeof(int));
    hipMalloc((void **)&device_matrix_product_result, MATRIX_DIMENSION * MATRIX_DIMENSION * sizeof(int));

    // Copy data from host buffers to device buffers.
    hipMemcpy(device_matrix_a, host_matrix_a, sizeof(int) * MATRIX_DIMENSION * MATRIX_DIMENSION, hipMemcpyKind::hipMemcpyHostToDevice);
    hipMemcpy(device_matrix_b, host_matrix_b, sizeof(int) * MATRIX_DIMENSION * MATRIX_DIMENSION, hipMemcpyKind::hipMemcpyHostToDevice);

    // Invoke the kernel to perform processing on the GPU.
    const dim3 thread_group_dim(32, 32, 1);
    const dim3 block_dim(MATRIX_DIMENSION / thread_group_dim.x, MATRIX_DIMENSION / thread_group_dim.y, 1);

    matrix_multiply<<<block_dim, thread_group_dim>>>(device_matrix_a, device_matrix_b, device_matrix_product_result);

    // Copy the result data from device to host memory.
    hipMemcpy(host_matrix_product_result, device_matrix_product_result, MATRIX_DIMENSION * MATRIX_DIMENSION * sizeof(int), hipMemcpyKind::hipMemcpyDeviceToHost);

    // Display results via console.
    printf("matrix A :: \n");
    for (int i = 0; i < MATRIX_DIMENSION; i++)
    {
        for (int j = 0; j < MATRIX_DIMENSION; j++)
        {
            printf("%d ", host_matrix_a[i + j * MATRIX_DIMENSION]);
        }
        printf("\n");
    }

    printf("\nmatrix B :: \n");
    for (int i = 0; i < MATRIX_DIMENSION; i++)
    {
        for (int j = 0; j < MATRIX_DIMENSION; j++)
        {
            printf("%d ", host_matrix_b[i + j * MATRIX_DIMENSION]);
        }
        printf("\n");
    }

    printf("\n matrix product :: \n");
    for (int i = 0; i < MATRIX_DIMENSION; i++)
    {
        for (int j = 0; j < MATRIX_DIMENSION; j++)
        {
            printf("%d ", host_matrix_product_result[i + j * MATRIX_DIMENSION]);
        }
        printf("\n");
    }

    printf("\n Expected resume :: \n");

    for (int i = 0; i < MATRIX_DIMENSION; i++)
    {
        for (int j = 0; j < MATRIX_DIMENSION; j++)
        {
            int sum = 0;
            for (int k = 0; k < MATRIX_DIMENSION; k++)
            {
                sum += host_matrix_a[k + i * MATRIX_DIMENSION] * host_matrix_b[k * MATRIX_DIMENSION + j];
            }

            printf("%d ", sum);
            if (sum != host_matrix_product_result[i + j * MATRIX_DIMENSION])
            {
                printf("ERROR at index %d %d.\n", i, j);
                return -1;
            }
        }
        printf("\n");
    }

    // Free allocated memory.
    free(host_matrix_a);
    free(host_matrix_b);
    free(host_matrix_product_result);

    hipFree(device_matrix_a);
    hipFree(device_matrix_b);
    hipFree(device_matrix_product_result);

    printf("%s", "All calculations were correct!\n");

    return 0;
}