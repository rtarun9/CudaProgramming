// Multi thread block hillis steele scan algorithm.
// This is a 2 phase algorithm. First, each block computes its local scan (using shared memory).
// Each block stores the value of its local final element to a differeant array. 
// This value is added to each element of threads block with index i + 1.
// NOTE : 
// Inorder to keep this algorithm limited to 3 passes, the MAXIMUM number of elements that can processed for now is 
// 1024 * 1024.


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define NUM_ELEMENTS 1024 * 1024
#define BLOCK_DIM 1024

// Use shared memory and find the scan (prefix sum) for a small portion of the overall input array.
__global__ void hillis_steele_local_scan(int* input_output_array, int* per_block_scan_output)
{
    __shared__ int smem[BLOCK_DIM];

    const int tx = threadIdx.x + blockIdx.x * blockDim.x;
    if (tx >= NUM_ELEMENTS)
    {
        return;
    }

    smem[threadIdx.x] = input_output_array[tx];
    __syncthreads();

    for (int stride = 1; stride < BLOCK_DIM; stride *= 2)
    {
        int val_to_add_to_smem = 0;
        if (threadIdx.x >= stride)
        {
            val_to_add_to_smem = smem[threadIdx.x - stride];
        }

        __syncthreads();

        smem[threadIdx.x] += val_to_add_to_smem;

        __syncthreads();
    }

    input_output_array[tx] = smem[threadIdx.x];

    if (per_block_scan_output && threadIdx.x == blockDim.x - 1)
    {
        per_block_scan_output[blockIdx.x] = smem[threadIdx.x];
    }
}

__global__ void add_array_with_per_block_scan_result(int* input_output_array, int* per_block_scan_output)
{
    const int tx = threadIdx.x + blockIdx.x * blockDim.x;

    if (blockIdx.x >= 1)
    {
        input_output_array[tx] += per_block_scan_output[blockIdx.x - 1];
    }
}

int main()
{
    constexpr size_t BYTES = NUM_ELEMENTS * sizeof(int);

    // Allocate and setup host side buffers.
    int* host_input_array = (int*)malloc(BYTES);
    int* host_output_array = (int*)malloc(BYTES);

    for (int i = 0; i < NUM_ELEMENTS; i++)
    {
        host_input_array[i] = 1;
    }

    // Allocate and setup device side buffers.
    int* device_input_output_array = nullptr;
    hipMalloc(&device_input_output_array, BYTES);
    hipMemcpy(device_input_output_array, host_input_array, BYTES, hipMemcpyKind::hipMemcpyHostToDevice);

    const dim3 block_dim(BLOCK_DIM, 1, 1);
    const dim3 grid_dim((NUM_ELEMENTS + BLOCK_DIM - 1) / BLOCK_DIM, 1, 1);

    int* device_per_block_scan_output_array = nullptr;
    hipMalloc(&device_per_block_scan_output_array, sizeof(int) * grid_dim.x);

    // Lauch kernel to compute per block local scan.
    hillis_steele_local_scan<<<grid_dim, block_dim>>>(device_input_output_array, device_per_block_scan_output_array);

    // Now find the scan value Of the per block scans done in the previous step :wow:
    // This DOES mean that number of blocks launched in previous step = number of elements in device_per_block_scan_output_array
    // must be less than or equal to maximum number of threads that can be processed in a block (1024 in this case).
    {
        const dim3 per_block_scan_grid_dim((grid_dim.x + BLOCK_DIM - 1) / BLOCK_DIM, 1, 1);
        hillis_steele_local_scan<<<per_block_scan_grid_dim, block_dim>>>(device_per_block_scan_output_array, nullptr);
    }

    // Now, add each element to the coresponding value of device_per_block_scan_output_array.
    add_array_with_per_block_scan_result<<<grid_dim, block_dim>>>(device_input_output_array, device_per_block_scan_output_array);

    hipMemcpy(host_output_array, device_input_output_array, BYTES, hipMemcpyKind::hipMemcpyDeviceToHost);

    bool success = true;
    int scan_result = 0;
    for (int i = 0; i < NUM_ELEMENTS; i++)
    {
        scan_result += host_input_array[i];
        if (scan_result != host_output_array[i])
        {
            printf("ERROR at index : %d. Got %d but expected %d.\n", i, host_output_array[i], scan_result);
            success = false;
        }
    }

    if (success)
    {
        printf("Algorithm was succesfull.");
    }

    free(host_input_array);
    free(host_output_array);

    hipFree(device_input_output_array);
    hipFree(device_per_block_scan_output_array);
}