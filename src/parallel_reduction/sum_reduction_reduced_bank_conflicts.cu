

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <random>

#define NUM_ELEMENTS (1 << 14)

__global__
void sum_reduction(int* input_array, int* output_array, int phase_count_start, int phase_count_end)
{
    __shared__ int smem[128 /* thread group dim*/];

    const int thread_id = threadIdx.x + blockDim.x * blockIdx.x;
    smem[threadIdx.x] = input_array[thread_id];
    __syncthreads();

    // Whats the issue with the previous (reduced divergence) algorithm?
    // Notice that the way we access memory is... non sequential.
    // In phase 1, you can index 0 accessing 0 and 1, index 1 accessing 2 and 3, etc.
    // This is fine for the most part. But, in later strides, you may have cases where
    // index 0 is accesssing 0 and 32, index 1 is accessing 64 and 96, etc.
    // the issue is there is a chance these elements (index 32 * X where X = 0, 1, 2, ..) lie in the same bank.
    // Think of memory bank like cache, where you have Y (say, 32 slots) and elements with address % Y == 0 lie in same bank.
    // In case of bank conflicts, the memory accesses become SEQUENTIAL.
    // How to prevent this? If memory access is linear for all threads (say index 0 accesses 0 and 8, index 1 access 1 adn 9, etc),
    // you have lot less bank conflicts.
    if (thread_id < NUM_ELEMENTS)
    {
        int stride = blockDim.x / 2;
        for (int i = phase_count_start; i <= phase_count_end; i++)
        {
            // Even though there is this if statement, divergence in the warp is still not too high.
            if (threadIdx.x < stride)
            {
                // the stride keeps reducing in this case.
                smem[threadIdx.x] += smem[threadIdx.x + stride];
                __syncthreads();
                stride /= 2;
            }
        }
    }

    if (threadIdx.x == 0)
    {
        printf("blockIdx.x %d has value %d\n", blockIdx.x, smem[threadIdx.x]);
        output_array[blockIdx.x] = smem[threadIdx.x];
    }
}

int main()
{
    int* host_array = (int*)malloc(NUM_ELEMENTS * sizeof(int));
    int* device_input_array = nullptr;
    int* device_output_array = nullptr;

    // Generate values for host array.
    for (int i = 0; i < NUM_ELEMENTS; i++)
    {
        host_array[i] = rand();
    }

    hipMalloc(&device_input_array, sizeof(int) * NUM_ELEMENTS);
    hipMalloc(&device_output_array, sizeof(int) * (int)ceilf(log2f(NUM_ELEMENTS)));

    hipMemcpy(device_input_array, host_array, sizeof(int) * NUM_ELEMENTS, hipMemcpyHostToDevice);

    const dim3 thread_group_dim = dim3(128, 1, 1);
    const dim3 thread_grid_dim = dim3((int)ceil(NUM_ELEMENTS / (float)thread_group_dim.x), 1, 1);

    if (thread_grid_dim.x > thread_group_dim.x)
    {
        printf("This would cause program to not work correctly, as the 2 kernel launches will have different block dim");
    }

    // For first pass, num_passes = log(thread_group_dim)
    const int num_passes = ceilf(log2f(thread_group_dim.x));

    // First pass, each thread group finds its sum and stores in the array.
    sum_reduction<<<thread_grid_dim, thread_group_dim>>>(device_input_array, device_output_array, 1, num_passes);
    // Second pass, find the sum of each thread block.
    sum_reduction<<<1, thread_grid_dim>>>(device_output_array, device_output_array, 1, (int)ceilf(log2f(thread_grid_dim.x)));

    hipDeviceSynchronize();

    size_t actual_res = 0;
    for (int i = 0; i < NUM_ELEMENTS; i++) actual_res += host_array[i];

    int computed_result = 0;
    hipMemcpy(&computed_result, device_output_array, sizeof(int), hipMemcpyKind::hipMemcpyDeviceToHost);

    printf("Parallel sum reduction with reduced bank conflicts.\n");
    printf("computed result %d\n", computed_result);
    printf("actual result %zd\n", actual_res);

    free(host_array);
    hipFree(device_input_array);
    hipFree(device_output_array);
}