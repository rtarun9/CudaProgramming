// Simple approach : Have 2 pass.
// One where each thread group finds the sum, and second where all thread groups find the collective sum.


#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <random>

#define NUM_ELEMENTS (1 << 14)

__global__
void sum_reduction(int* input_array, int* output_array, int phase_count_start, int phase_count_end)
{
    __shared__ int smem[128 /* thread group dim*/];

    const int thread_id = threadIdx.x + blockDim.x * blockIdx.x;
    smem[threadIdx.x] = input_array[thread_id];
    __syncthreads();

    if (thread_id < NUM_ELEMENTS)
    {
        for (int i = phase_count_start; i <= phase_count_end; i++)
        {
            const int stride = 1 << i;

            if (thread_id % stride == 0)
            {
                smem[threadIdx.x] += smem[threadIdx.x + stride / 2];
            }
            __syncthreads();
        }
    }

    if (threadIdx.x == 0)
    {
        printf("blockIdx.x %d has value %d\n", blockIdx.x, smem[threadIdx.x]);
        output_array[blockIdx.x] = smem[threadIdx.x];
    }
}

int main()
{
    int* host_array = (int*)malloc(NUM_ELEMENTS * sizeof(int));
    int* device_input_array = nullptr;
    int* device_output_array = nullptr;

    // Generate values for host array.
    for (int i = 0; i < NUM_ELEMENTS; i++)
    {
        host_array[i] = 1;
    }

    hipMalloc(&device_input_array, sizeof(int) * NUM_ELEMENTS);
    hipMalloc(&device_output_array, sizeof(int) * (int)ceilf(log2f(NUM_ELEMENTS)));

    hipMemcpy(device_input_array, host_array, sizeof(int) * NUM_ELEMENTS, hipMemcpyHostToDevice);

    const dim3 thread_group_dim = dim3(128, 1, 1);
    const dim3 thread_grid_dim = dim3((int)ceil(NUM_ELEMENTS / (float)thread_group_dim.x), 1, 1);

    if (thread_grid_dim.x > thread_group_dim.x)
    {
        printf("This would cause program to not work correctly, as the 2 kernel launches will have different block dim");
    }

    // For first pass, num_passes = log(thread_group_dim)
    const int num_passes = ceilf(log2f(thread_group_dim.x));

    // First pass, each thread group finds its sum and stores in the array.
    sum_reduction<<<thread_grid_dim, thread_group_dim>>>(device_input_array, device_output_array, 1, num_passes);
    // Second pass, find the sum of each thread block.
    sum_reduction<<<1, thread_grid_dim>>>(device_output_array, device_output_array, 1, (int)ceilf(log2f(thread_grid_dim.x)));

    hipDeviceSynchronize();

    size_t actual_res = 0;
    for (int i = 0; i < NUM_ELEMENTS; i++) actual_res += host_array[i];

    int computed_result = 0;
    hipMemcpy(&computed_result, device_output_array, sizeof(int), hipMemcpyKind::hipMemcpyDeviceToHost);

    printf("computed result %d\n", computed_result);
    printf("actual result %zd\n", actual_res);

    free(host_array);
    hipFree(device_input_array);
    hipFree(device_output_array);
}