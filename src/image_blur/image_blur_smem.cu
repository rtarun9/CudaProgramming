#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#define STB_IMAGE_IMPLEMENTATION
#include "../../external/stb_image.h"

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "../../external/stb_image_write.h"


// Now, whats the issue with the naive solution?
// Too much global memory accesses.
// Okay.. so shared memory is potential solution
// But how is data to be stored within a group? since pixels at the border of 2 thread groups will face a issue...
// Solution : When loading data into shared memory, in each axis load additional filter_width / 2 elements in each dim.
// The corner elements will load these into shared memory. With this, a entire thread block can perform convolution
// straight from shared memory.
#define BLOCK_DIM 8
#define FILTER_DIM 5
#define SMEM_ARRAY_DIM (BLOCK_DIM + FILTER_DIM / 2)

__global__ void shared_mem_blur(const unsigned char* input_image, unsigned char* output_image, int width, int height)
{
    __shared__ int smem_pixel_values[SMEM_ARRAY_DIM][SMEM_ARRAY_DIM];

    const int t_x = threadIdx.x + blockIdx.x * blockDim.x;
    const int t_y = threadIdx.y + blockIdx.y * blockDim.y;

    // Now, load into shared mem. Keep in mind that border pixels need some extra work to do.
    smem_pixel_values[threadIdx.x][threadIdx.y] = input_image[t_x + t_y * width];
    __syncthreads();

    // Now check for corner condition.
    if (threadIdx.x == 0)
    {
        // Load data to the left.
        for (int i = 1; i <= FILTER_DIM / 2; i--) 
        {
            smem_pixel_values[threadIdx.x - i][threadIdx.y] = input_image[t_x - i + (t_y * width)];
        }
    }

    if (threadIdx.x == blockDim.x - 1)
    {
        // Load data to the right.
        for (int i = 0; i <= FILTER_DIM / 2; i++) 
        {
            smem_pixel_values[threadIdx.x + i][threadIdx.y] = input_image[t_x + i + (t_y * width)];
        }
    }

    if (threadIdx.y == 0)
    {
        // Load data up.
        for (int i = 1; i <= FILTER_DIM / 2; i++) 
        {
            smem_pixel_values[threadIdx.x][threadIdx.y - i] = input_image[t_x + ((t_y  - i) * width)];
        }
    }

    if (threadIdx.y == blockDim.y - 1)
    {
        // Load data up.
        for (int i = 1; i <= FILTER_DIM / 2; i++) 
        {
            smem_pixel_values[threadIdx.x][threadIdx.y + i] = input_image[t_x + ((t_y  + i) * width)];
        }
    }

    const size_t pixel_index = t_x + t_y * width;

    float pixel_sum = 0.0f;
    for (int i = -2; i <= 2; i++)
    {
        for (int j = -2; j <= 2; j++)
        {
            pixel_sum += smem_pixel_values[threadIdx.x + j][threadIdx.y + i];
        }
    }

    output_image[pixel_index] = (unsigned char)(pixel_sum / 25.0f);

    return;
}

int main()
{
    // First, read the source image and extract relavant data.
    int width = 0;
    int height = 0;
    unsigned char* h_input_image_data = stbi_load("../../assets/images/test_image_grayscale.png", &width, &height, nullptr, 1);

    printf("Image width and height : %d %d\n", width, height);

    // Allocate memory for the output (host) data and input and output (device) data.
    const size_t GRAY_SCALE_IMAGE_BYTES = sizeof(unsigned char) * width * height;

    unsigned char* h_output_image_data = (unsigned char*)malloc(GRAY_SCALE_IMAGE_BYTES);

    unsigned char* d_input_image_data = nullptr;
    unsigned char* d_output_image_data = nullptr;

    hipMalloc(&d_input_image_data, GRAY_SCALE_IMAGE_BYTES);
    hipMalloc(&d_output_image_data, GRAY_SCALE_IMAGE_BYTES);

    hipMemcpy(d_input_image_data, h_input_image_data, GRAY_SCALE_IMAGE_BYTES, hipMemcpyKind::hipMemcpyHostToDevice);

    // Launch kernel.
    // Each thread block will be of 16 x 16 threads. Based on input image, find the number of blocks to launch.
    const dim3 block_dim = dim3(BLOCK_DIM, BLOCK_DIM, 1);
    const dim3 grid_dim = dim3((width + block_dim.x - 1) / block_dim.x, (height + block_dim.y - 1) / block_dim.y, 1);

    printf("Block dim :: %d %d %d\n", block_dim.x, block_dim.y, block_dim.z);
    printf("Grid dim :: %d %d %d\n", grid_dim.x, grid_dim.y, grid_dim.z);

    // shared mem blur.
    {
        shared_mem_blur<<<grid_dim, block_dim>>>(d_input_image_data, d_output_image_data, width, height);

        // Copy output to host memory.
        hipMemcpy(h_output_image_data, d_output_image_data, GRAY_SCALE_IMAGE_BYTES, hipMemcpyKind::hipMemcpyDeviceToHost);

        // Write output in image format (with file name : output_image_grayscale.png).
        const size_t output_image_row_stride = sizeof(unsigned char) * 1 * width;
        if (stbi_write_png("../../assets/images/shared_mem_blur.png", width, height, 1, h_output_image_data, output_image_row_stride))
        {
            printf("Successfully wrote output image to ../assets/images/shared_mem_blur.png");
        }
        else
        {
            printf("Failed to write to output image");
        }
    }

    stbi_image_free(h_input_image_data);
    free(h_output_image_data);

    hipFree(d_input_image_data);
    hipFree(d_output_image_data);
}