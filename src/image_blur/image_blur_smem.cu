#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#define STB_IMAGE_IMPLEMENTATION
#include "../../external/stb_image.h"

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "../../external/stb_image_write.h"


// Now, whats the issue with the naive solution?
// Too much global memory accesses.
// Okay.. so shared memory is potential solution
// But how is data to be stored within a group? since pixels at the border of 2 thread groups will face a issue...
// Solution : When loading data into shared memory, for corner pixels each axis load additional filter_width / 2 elements in each dim.
// The corner elements will load these into shared memory. With this, a entire thread block can perform convolution
// straight from shared memory.
#define BLOCK_DIM 8
#define FILTER_DIM 5
#define SMEM_ARRAY_DIM (BLOCK_DIM + FILTER_DIM)

__global__ void shared_mem_blur(const unsigned char* input_image, unsigned char* output_image, int width, int height)
{
    __shared__ int smem_pixel_values[SMEM_ARRAY_DIM][SMEM_ARRAY_DIM];

    // Input image pixel that maps to center of stencil.
    const int t_x =  FILTER_DIM / 2 + threadIdx.x + blockIdx.x * blockDim.x;
    const int t_y = FILTER_DIM / 2 + threadIdx.y + blockIdx.y * blockDim.y;

    // which output image pixel the thread maps to.
    const int output_t_x =  threadIdx.x + blockIdx.x * blockDim.x;
    const int output_t_y =  threadIdx.y + blockIdx.y * blockDim.y;

    const int input_image_width = width + FILTER_DIM;

    // Because of padding, there is no longer a straight 1-1 mapping from thread to image, as padding has to be accounted for.
    // Also keep in mind that input and output image are no longer the same dimension, as output has NO padding.
    const int smem_index_x = threadIdx.x + FILTER_DIM / 2;
    const int smem_index_y = threadIdx.y + FILTER_DIM / 2;

    // Now, load into shared mem. Keep in mind that border pixels need some extra work to do.
    // There is a bit of strange math here. The shared mem for this element is represented by threadIdx.x + FILTER_DIM / 2
    // This is to accomidate the extra data that is being loaded.
    smem_pixel_values[smem_index_x][smem_index_y] = input_image[t_x + t_y * input_image_width];
    __syncthreads();

    // For border threads (in the block), load the additional data into shared memory.
    if (threadIdx.x == 0)
    {
        for (int i = 1; i <= FILTER_DIM / 2; i++)
        {
            smem_pixel_values[smem_index_x - i][smem_index_y] = input_image[t_x - i + t_y * input_image_width];
        }
    }

    if (threadIdx.x == blockDim.x - 1)
    {
        for (int i = 1; i <= FILTER_DIM / 2; i++)
        {
            smem_pixel_values[smem_index_x + i][smem_index_y] = input_image[t_x + i + t_y * input_image_width];
        }

    }

    if (threadIdx.y == blockDim.x - 1)
    {
        for (int i = 1; i <= FILTER_DIM / 2; i++)
        {
            smem_pixel_values[smem_index_x][smem_index_y + i] = input_image[t_x +  (t_y + i) * input_image_width];
        }

    }

    if (threadIdx.y == 0)
    {
        for (int i = 1; i <= FILTER_DIM / 2; i++)
        {
            smem_pixel_values[smem_index_x][smem_index_y - i] = input_image[t_x +  (t_y - i) * input_image_width];
        }
    }

    __syncthreads();

    // NOTE : Remove this once a suitable solution is found! Special edge case for teh corner pixels (4 per block)
    const size_t pixel_index = output_t_x+ output_t_y * width;

    float pixel_sum = 0.0f;
    for (int i = -FILTER_DIM/2; i <= FILTER_DIM/2; i++)
    {
        for (int j = -FILTER_DIM/2; j <= FILTER_DIM/2; j++)
        {
            pixel_sum += smem_pixel_values[smem_index_x + i][smem_index_y+ j];
        }
    }

    output_image[pixel_index] = (unsigned char)(pixel_sum / (float)(FILTER_DIM * FILTER_DIM));

    return;
}

int main()
{
    // First, read the source image and extract relavant data.
    int width = 0;
    int height = 0;
    unsigned char* h_input_image_data = stbi_load("../../assets/images/test_image_grayscale.png", &width, &height, nullptr, 1);

    // Note : While input image will end up being padded, output image width and height will match the below values.
    printf("Image width and height : %d %d\n", width, height);

    unsigned char* h_padded_input_image_data = (unsigned char*)calloc((width + FILTER_DIM) * (height + FILTER_DIM), sizeof(unsigned char));
    for (int i = 0; i < height; i++)
    {
        for (int j = 0; j < width; j++)
        {
            h_padded_input_image_data[j + (FILTER_DIM / 2) + (i + FILTER_DIM / 2)  * (width + FILTER_DIM)] = h_input_image_data[j + i * width];
        }
    }

    // Allocate memory for the output (host) data and input and output (device) data.
    const size_t UNPADDED_GRAY_SCALE_IMAGE_BYTES = sizeof(unsigned char) * width * height;
    const size_t PADDED_GRAY_SCALE_IMAGE_BYTES = sizeof(unsigned char) * (width + FILTER_DIM) * (height+FILTER_DIM);

    unsigned char* h_output_image_data = (unsigned char*)malloc(UNPADDED_GRAY_SCALE_IMAGE_BYTES);

    unsigned char* d_input_image_data = nullptr;
    unsigned char* d_output_image_data = nullptr;

    hipMalloc(&d_input_image_data, PADDED_GRAY_SCALE_IMAGE_BYTES);
    hipMalloc(&d_output_image_data, UNPADDED_GRAY_SCALE_IMAGE_BYTES);

    hipMemcpy(d_input_image_data, h_padded_input_image_data, PADDED_GRAY_SCALE_IMAGE_BYTES, hipMemcpyKind::hipMemcpyHostToDevice);

    // Launch kernel.
    // Each thread block will be of 8 x 8 threads. Based on input image, find the number of blocks to launch.
    const dim3 block_dim = dim3(BLOCK_DIM, BLOCK_DIM, 1);
    const dim3 grid_dim = dim3((width + block_dim.x - 1) / block_dim.x, (height + block_dim.y - 1) / block_dim.y, 1);

    printf("Block dim :: %d %d %d\n", block_dim.x, block_dim.y, block_dim.z);
    printf("Grid dim :: %d %d %d\n", grid_dim.x, grid_dim.y, grid_dim.z);

    // shared mem blur.
    {
        shared_mem_blur<<<grid_dim, block_dim>>>(d_input_image_data, d_output_image_data, width, height);

        // Copy output to host memory.
        hipMemcpy(h_output_image_data, d_output_image_data, UNPADDED_GRAY_SCALE_IMAGE_BYTES, hipMemcpyKind::hipMemcpyDeviceToHost);

        // Write output in image format (with file name : output_image_grayscale.png).
        const size_t output_image_row_stride = sizeof(unsigned char) * 1 * width;
        if (stbi_write_png("../../assets/images/shared_mem_blur.png", width, height, 1, h_output_image_data, output_image_row_stride))
        {
            printf("Successfully wrote output image to ../assets/images/shared_mem_blur.png");
        }
        else
        {
            printf("Failed to write to output image");
        }
    }

    stbi_image_free(h_input_image_data);
    free(h_output_image_data);
    free(h_padded_input_image_data);

    hipFree(d_input_image_data);
    hipFree(d_output_image_data);
}